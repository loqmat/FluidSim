#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "fluids/cuda_setup.hpp"
#include "fluids/setup.hpp"
#include "fluids/graphics.hpp"
#include "fluids/camera.hpp"
#include "fluids/cuda_buffer.hpp"
#include "fluids/cuda_uniform_buffer.hpp"
#include "fluids/gen_sphere.hpp"

#include <vector>
#include <iostream>

using namespace Fluids;

#define VERTEX_COUNT 65536
#define THREAD_COUNT 512
#define GRAVITY_POINTS 4	

void createFlatShader(Shader& shad);

__global__ void gravity(float dt, core::vec4* gravpts, core::vec4* velocity, core::vec4* position) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float dx[GRAVITY_POINTS],
		  dy[GRAVITY_POINTS],
		  dz[GRAVITY_POINTS],
		  mg[GRAVITY_POINTS];

	for( int j=0;j<GRAVITY_POINTS;j++ ) {
		dx[j] = gravpts[j].x - position[i].x;
		dy[j] = gravpts[j].y - position[i].y;
		dz[j] = gravpts[j].z - position[i].z;

		mg[j] = (dx[j]*dx[j] + dy[j]*dy[j] + dz[j]*dz[j]) * gravpts[j].w;

		velocity[i].x += dx[j] / mg[j] * dt;
		velocity[i].y += dy[j] / mg[j] * dt;
		velocity[i].z += dz[j] / mg[j] * dt;
	}
	
	position[i].x += velocity[i].x * dt;
	position[i].y += velocity[i].y * dt;
	position[i].z += velocity[i].z * dt;
}

struct GlobalData {
	static bool rightMousePressed;

	static double xPosition;
	static double yPosition;

	static double xOldPosition;
	static double yOldPosition;

	static double xDeltaPosition;
	static double yDeltaPosition;

	static double xScroll;
	static double yScroll;

	static void UpdateMouse() {
		xDeltaPosition = xPosition - xOldPosition;
		xOldPosition = xPosition;

		yDeltaPosition = yPosition - yOldPosition;
		yOldPosition = yPosition;

		xScroll = 0.0;
		yScroll = 0.0;
	}
};

bool GlobalData::rightMousePressed = false;
double GlobalData::xPosition = 0;
double GlobalData::yPosition = 0;
double GlobalData::xOldPosition = 0;
double GlobalData::yOldPosition = 0;
double GlobalData::xDeltaPosition = 0;
double GlobalData::yDeltaPosition = 0;
double GlobalData::xScroll = 0;
double GlobalData::yScroll = 0;

void mousePositionCallback(GLFWwindow* window, double xpos, double ypos) {
	GlobalData::xPosition = xpos;
	GlobalData::yPosition = ypos;
}
void mouseButtonCallback(GLFWwindow* window, int button, int action, int mods) {
	if ( button == GLFW_MOUSE_BUTTON_RIGHT ) {
		if (action == GLFW_PRESS) {
			GlobalData::rightMousePressed = true;
		} else if (action == GLFW_RELEASE) {
			GlobalData::rightMousePressed = false;
		}
	}
}
void mouseScrollCallback(GLFWwindow* window, double xoffset, double yoffset) {
	GlobalData::xScroll = xoffset;
	GlobalData::yScroll = yoffset;
}

void run(const std::vector<std::string>& args) {
//--------------------------------------------------------------------------------------------------
// Initialize Components
//--------------------------------------------------------------------------------------------------
	InitGLFW glfw;
	Window mainWindow("Fluid Simulator", 1600, 900);
	glfwMakeContextCurrent(mainWindow);

	InitGLEW glew;
	glew.initialize();

	glfwSwapInterval(1);

	glfwSetCursorPosCallback(mainWindow, mousePositionCallback);
	glfwSetMouseButtonCallback(mainWindow, mouseButtonCallback);
	glfwSetScrollCallback(mainWindow, mouseScrollCallback);

	CUDA cuda;
	cuda.Setup();
	cuda.GLSetup();

	glEnable( GL_PROGRAM_POINT_SIZE );

//--------------------------------------------------------------------------------------------------
// Discover Rendering Restrictions
//--------------------------------------------------------------------------------------------------

	GLint draw_iterations;
	GLint max_uniform_buffer_range;
	GLint max_uniform_buffer_units;

	glGetIntegerv(GL_MAX_VERTEX_UNIFORM_COMPONENTS, &max_uniform_buffer_range);

	draw_iterations = 4 * VERTEX_COUNT / max_uniform_buffer_range;
	max_uniform_buffer_units = max_uniform_buffer_range / 4;

	std::cerr << "Max Number of Vec4s  : " << max_uniform_buffer_units << std::endl;
	std::cerr << "Max Number of Floats : " << max_uniform_buffer_range << std::endl;
	std::cerr << "Number of Draw Calls : " << draw_iterations << std::endl;

//--------------------------------------------------------------------------------------------------
// Generate Draw Buffer
//--------------------------------------------------------------------------------------------------

	GLuint vertex_buffer;
	glGenBuffers(1, &vertex_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, vertex_buffer);
	{
		core::vec4* position = new core::vec4[max_uniform_buffer_units];
		for (int i=0;i<max_uniform_buffer_units;i++)
			position[i].w = 1.0f;
		glBufferData(GL_ARRAY_BUFFER, sizeof(core::vec4) * max_uniform_buffer_units, position, GL_STATIC_DRAW);
		delete[] position;
	}

	int draw_vertex_count, // number of vertices in the sphere
		draw_face_count; // number of faces in the sphere
	GLuint array_buffer;
	GLuint index_buffer;

	generateSphere( draw_vertex_count, draw_face_count, array_buffer, index_buffer, 1.0f, 4 );

//--------------------------------------------------------------------------------------------------
// Initialize Position and Velocity Buffers
//--------------------------------------------------------------------------------------------------

	srand(time(NULL));

	UniformBuffer<core::vec4> matrix_data(sizeof(core::vec4) * VERTEX_COUNT);
	{
		glBindBuffer(GL_COPY_WRITE_BUFFER, matrix_data.handleGL());
		float position_range = 64.0f;
		core::vec4* position = new core::vec4[VERTEX_COUNT];
		for (int i=0;i<VERTEX_COUNT;i++) {
			position[i].x = position_range * ((float)rand() / RAND_MAX - 0.5f);
			position[i].y = position_range * ((float)rand() / RAND_MAX - 0.5f);
			position[i].z = position_range * ((float)rand() / RAND_MAX - 0.5f);
			position[i].w = 0.0f;
		}
		glBufferSubData(GL_COPY_WRITE_BUFFER, 0, sizeof(core::vec4) * VERTEX_COUNT, (void*)position);
		delete[] position;
		glBindBuffer(GL_COPY_WRITE_BUFFER, 0);
	}

	CUDABuffer<core::vec4> gravity_data(GRAVITY_POINTS);
	{
		float gravity_range = 128.0f;
		float strength_range = 0.75f;
		core::vec4* gravpts = new core::vec4[GRAVITY_POINTS];
		for( int i=0;i<GRAVITY_POINTS;i++ ) {
			gravpts[i].x = gravity_range * ((float)rand() / RAND_MAX - 0.5f);
			gravpts[i].y = gravity_range * ((float)rand() / RAND_MAX - 0.5f);
			gravpts[i].z = gravity_range * ((float)rand() / RAND_MAX - 0.5f);
			gravpts[i].w = strength_range * rand() / RAND_MAX + 0.25f;
		}
		gravity_data.upload(gravpts);
		delete[] gravpts;
	}

	CUDABuffer<core::vec4> input_data(VERTEX_COUNT);
	{
		float velocity_range = 16.0f;
		core::vec4* velocity = new core::vec4[VERTEX_COUNT];
		for( int i=0;i<VERTEX_COUNT;i++ ) {
			velocity[i].x = velocity_range * ((float)rand() / RAND_MAX - 0.5f);
			velocity[i].y = velocity_range * ((float)rand() / RAND_MAX - 0.5f);
			velocity[i].z = velocity_range * ((float)rand() / RAND_MAX - 0.5f);
			velocity[i].w = 0.0f;
		}
		input_data.upload(velocity);
		delete[] velocity;
	}

//--------------------------------------------------------------------------------------------------
// OpenGL Shaders and Uniforms
//--------------------------------------------------------------------------------------------------

	Camera main_camera;
	main_camera.arm_length = 128.0f;
	main_camera.rise = -1.0f;

	Shader shader_flat;
	GLuint shader_proj_view = 0;
	GLuint shader_light_direc = 0;

	createFlatShader(shader_flat);
	glUseProgram(shader_flat);
	shader_proj_view = glGetUniformLocation(shader_flat, "u_projection_view");
	shader_light_direc = glGetUniformLocation(shader_flat, "u_light_direction");

	GLuint modelview_index = glGetUniformBlockIndex(shader_flat, "ModelView");   
	glUniformBlockBinding(shader_flat, modelview_index, 0);

	if ( glGetError() != GL_NO_ERROR ) 
		throw "Got OpenGL Error during Setup!";

//--------------------------------------------------------------------------------------------------
// MAIN LOOP
//--------------------------------------------------------------------------------------------------
	double _current_time = glfwGetTime();
	double _delta_time = 0.0;
	double _fps = 0.0f;

	while (!glfwWindowShouldClose(mainWindow))
	{

	//----------------------------------------------------------------------------------------------
	// CUDA Segment
	//----------------------------------------------------------------------------------------------
		{
			matrix_data.bindCUDA();
			gravity<<<VERTEX_COUNT/THREAD_COUNT,THREAD_COUNT>>>(
				(float)_delta_time,						// frame time
				(core::vec4*)gravity_data,				// gravity
				(core::vec4*)input_data,				// velocity
				(core::vec4*)matrix_data);	// position
			checkCUDAResult();
			matrix_data.unbindCUDA();
		}

	//----------------------------------------------------------------------------------------------
	// OpenGL Segment
	//----------------------------------------------------------------------------------------------
		{
			int width, height;
			glfwGetFramebufferSize(mainWindow, &width, &height);

			if ( GlobalData::rightMousePressed ) {
				main_camera.angle -= 0.8f * deg2rad * (float)GlobalData::xDeltaPosition;
				main_camera.rise = (float)std::min( 80.0 * deg2rad, 
										  std::max( -80.0 * deg2rad, 
										  (double)main_camera.rise - 0.3 * deg2rad * GlobalData::yDeltaPosition ) );
			}
			if ( GlobalData::yScroll < 0 )
				main_camera.arm_length = std::min(1024.0f, main_camera.arm_length * (float)std::pow(1.1f, -GlobalData::yScroll));
			else if ( GlobalData::yScroll > 0 )
				main_camera.arm_length = std::max(16.0f, main_camera.arm_length * (float)std::pow(0.9f, GlobalData::yScroll));

			{
				core::mat4 data;
				main_camera.fillMatrix((float)width/height, data);
				glUniformMatrix4fv(shader_proj_view, 1, false, (float*)&data);
				glUniform3f(shader_light_direc, 0,1,0);
			}

			glViewport(0, 0, width, height);
			glClearColor(1,0,0,1);
			glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT);

			glBindBuffer(GL_ARRAY_BUFFER, array_buffer);
			glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, index_buffer);

			glEnableVertexAttribArray(0);
			glEnableVertexAttribArray(1);
			glEnableVertexAttribArray(2);
			glEnableVertexAttribArray(3);

			glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (void*)(std::size_t)(draw_vertex_count * sizeof(float) * 0));
			glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, (void*)(std::size_t)(draw_vertex_count * sizeof(float) * 3));
			glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, (void*)(std::size_t)(draw_vertex_count * sizeof(float) * 6));
			glVertexAttribPointer(3, 3, GL_FLOAT, GL_FALSE, 0, (void*)(std::size_t)(draw_vertex_count * sizeof(float) * 8));

			for ( int iteration=0;iteration<draw_iterations;iteration++ ) {
				matrix_data.bindGL(0, iteration * max_uniform_buffer_range, max_uniform_buffer_range);
				glDrawElementsInstanced(GL_TRIANGLES, 3 * draw_face_count, GL_UNSIGNED_BYTE, 0, max_uniform_buffer_units);
			}

			glFinish();
		}

	//----------------------------------------------------------------------------------------------
	// Finish Frame
	//----------------------------------------------------------------------------------------------
		{
			glfwSwapBuffers(mainWindow);
			GlobalData::UpdateMouse();
			glfwPollEvents();

			double ct = glfwGetTime();
			
			_delta_time = ct - _current_time;
			_current_time = ct;
			_fps = 0.9 * _fps + 0.1 * (1.0 / _delta_time);
		}

	}
}

int main(int argv, char** argc) {
	try {
		std::vector<std::string> data(argv);
		for (int i = 0; i < argv; i++)
			data[i] = argc[i];
		run(data);
	} catch (const char* err) {
		std::cerr << "Got an error: \"" << err << "\"" << std::endl;
	}
	return 0;
}

void createFlatShader(Shader& shad) {
	shad.vertex =
		"#version 330\r\n"

		"layout (std140) uniform ModelView"
		"{"
		"	vec4 u_model[1024];"
		"};"
		"uniform mat4 u_projection_view;"

		"in vec3 in_position;"
		"in vec3 in_normal;"
		"in vec2 in_uv;"
		"in vec3 in_color;"

		"out vec2 v_uv;"
		"out vec3 v_normal;"
		"out vec3 v_color;"

		"void main() {"
		"	vec4 local = u_model[gl_InstanceID];"
		"   gl_Position = u_projection_view * (local + vec4(in_position, 1));"
		"	v_uv = in_uv;"
		"	v_normal = in_normal;"
		"	v_color = in_color;"
		"}";
		
	shad.fragment =
		"#version 330\r\n"

		"uniform vec3 u_light_direction;"

		"in vec2 v_uv;"
		"in vec3 v_normal;"
		"in vec3 v_color;"

		"out vec4 out_color;"

		"void main() {"
		"	float lighting = 0.8 * clamp(dot(u_light_direction, v_normal) + 0.2, 0, 1) + 0.2;"
		"	out_color = vec4(lighting * v_color,1);"
		"}";
	shad.link();
}